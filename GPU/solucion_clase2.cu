#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream> 
#include <sstream>
#include <string> 
#include <memory>

#include <hip/hip_runtime.h>

__global__ void kernel(int N, int* A, int* B, int* C){
	int tId = blockDim.x * blockIdx.x + threadIdx.x;
	if (tId < N) {
		C[tId] = A[i] + B[i];
	}
	
}

int main(int argc, char **argv){	
	
	hipError_t err = hipSuccess;	
	int N = 256;      
	int Nblocks = 1;
	int Nthreads = 256;
	int* d_A = NULL;
	int* d_B = NULL;
	int* d_C = NULL;
	int* A = new int[N];
	int* B = new int[N];
	int* C = new int[N];
	size_t size = N * sizeof(int);
	
	for(int i = 0; i < N ;i++){
			 A[i] = 1;
			 B[i] = 2;
	}		

	err = hipMalloc((void **)&d_A, size);
	if (err != hipSuccess)
	{
			fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
	}
	err = hipMalloc((void **)&d_B, size);
	if (err != hipSuccess)
	{
			fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
	}
	err = hipMalloc((void **)&d_C, size);
	if (err != hipSuccess)
	{
			fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
	}

	err = hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
	if (err != hipSuccess)
	{
			fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
	}
	err = hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
	if (err != hipSuccess)
	{
			fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
	}
	
	kernel<<<Nblocks, Nthreads>>>(N, d_A, d_B,d_C);
	hipDeviceSynchronize();
	err = hipGetLastError();
if (err != hipSuccess) {
    fprintf(stderr, "Failed to launch kernel (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
}

  err = hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }
	bool flag = true;
	for(int i = 0; i < N ;i++){		
		if( C[i] != A[i] + B[i]){
			flag = false;
		}				
	}
	if(flag){
		std::cout << "Vector Add Ok :D" << std::endl;
	}else{
		std::cout << "Oh no, Error detected D:"<< std::endl;
	}


	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	delete[] A;
	delete[] B;
	delete[] C;

	return 0;
}
#include <iostream>
#include <fstream> 
#include <sstream>
#include <string> 
#include <memory>

#include <hip/hip_runtime.h>

__global__ void kernel(int N, int* A, int* B, int* C){
	int tId = blockDim.x * blockIdx.x + threadIdx.x;
	/*
	Código
	*/	
	
}

int main(int argc, char **argv){	
	
	hipError_t err = hipSuccess;	
	int N = 256;      
	int Nblocks = 1;
	int Nthreads = 256;
	int* d_A = NULL;
	int* d_B = NULL;
	int* d_C = NULL;
	int* A = new int[N];
	int* B = new int[N];
	int* C = new int[N];
	size_t size = N * sizeof(int);
	
	for(int i = 0; i < N ;i++){
			 A[i] = 1;
			 B[i] = 2;
	}		
	
	/*
	Código
	*/	
	kernel<<<Nblocks, Nthreads>>>(N, d_A, d_B,d_C);
	hipDeviceSynchronize();
	err = hipGetLastError();
	if (err != hipSuccess) {
	    fprintf(stderr, "Failed to launch kernel (error code %s)!\n", hipGetErrorString(err));
	    exit(EXIT_FAILURE);
	}

  /*
	Código
	*/	

	bool flag = true;
	for(int i = 0; i < N ;i++){		
		if( C[i] != A[i] + B[i]){
			flag = false;
		}				
	}

	if(flag){
		std::cout << "Vector Add Ok :D" << std::endl;
	}else{
		std::cout << "Oh no, Error detected D:"<< std::endl;
	}

	delete[] A;
	delete[] B;
	delete[] C;

	return 0;
}